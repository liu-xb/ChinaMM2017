#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/util/math_functions.hpp"
#include "caffe/layers/p_power1_layer.hpp"

namespace caffe {

template <typename Dtype>
  __global__ void PPower1Forward (const int count, const Dtype* bottom, const Dtype p, Dtype* top)
  {
    CUDA_KERNEL_LOOP(index, count)
    {
      if (bottom[index] > 0)
      {
        top[index] = pow(bottom[index], p);  
      }
    }
  }
template <typename Dtype>
void PPower1Layer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype p = this->blobs_[0]->cpu_data()[0];
  PPower1Forward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, bottom_data, p, top_data);
  CUDA_POST_KERNEL_CHECK;
  if(iter_ == -1)
  {
    temp_diff_.Reshape(bottom[0]->shape());
    LOG(INFO)<<"p = "<<p;
  }
  ++iter_;
  if (iter_ == 500)
  {
    LOG(INFO)<<"p = "<<p<<'\n';
    iter_ = 0;
  }
}

template <typename Dtype>
__global__ void PPOwer1Backward(const int count, Dtype* top_data, const Dtype* bottom_data, const Dtype* top_diff, Dtype* bottom_diff, const Dtype p)
{
  CUDA_KERNEL_LOOP(index, count)
  {
    bottom_diff[index] = top_diff[index] * p * top_data[index] / bottom_data[index];
  }
}

template <typename Dtype>
__global__ void PGradient(const int count, Dtype* top_data, const Dtype* bottom_data, const Dtype* top_diff, Dtype* temp_diff)
{
  CUDA_KERNEL_LOOP(index, count)
  {
    if (bottom_data[index] > 0)
    {
      temp_diff[index] = top_diff[index] * top_data[index] * log(bottom_data[index]);
      top_data[index] = 1;
    }
    else
    {
      temp_diff[index] = 0.0;
      top_data[index] = 0.0;
    }
  }
}

template <typename Dtype>
void PPower1Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff(); // diff_p is temporarily saved in top_diff, so top_diff is changed.
  Dtype* top_data = top[0]->mutable_gpu_data(); // number of non-zero diff_p is saved in top_data, so top_data is changed.
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* temp_diff = temp_diff_.mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype diff_p(0);
  Dtype diff_p_num(0);
  Dtype p = this->blobs_[0]->cpu_data()[0];
  if (propagate_down[0]) {
    PPOwer1Backward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data, bottom_data, top_diff, bottom_diff, p);
  }
  CUDA_POST_KERNEL_CHECK;
  PGradient<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data, bottom_data, top_diff, temp_diff);
  CUDA_POST_KERNEL_CHECK;
  caffe_gpu_dot<Dtype>(count, temp_diff, top_data, &diff_p);
  // caffe_gpu_asum<Dtype>(count, temp_diff, &diff_p);
  // caffe_gpu_asum<Dtype>(count, top_data, &diff_p_num);
  // this->blobs_[0]->mutable_cpu_diff()[0] = diff_p;// / (diff_p_num + 1e-5);
  // LOG(INFO)<<diff_p<<"aa";
  // diff_p = 0;
  // for (int i = 0; i < count; ++i) 
  // {
  //     if (bottom[0]->cpu_data()[i] > 0)
  //     {
  //       Dtype t = top[0]->cpu_diff()[i] * top[0]->cpu_data()[i] * log(bottom[0]->cpu_data()[i]);
  //       diff_p += t;
  //       // LOG(INFO)<<t<<"  adf   "<<temp_diff_.cpu_data()[i];
  //       // ++diff_p_num;
  //     }
  // }
  // LOG(INFO)<<diff_p<<"bb";
  this->blobs_[0]->mutable_cpu_diff()[0] = diff_p; // /diff_p_num;
}
INSTANTIATE_LAYER_GPU_FUNCS(PPower1Layer);
}  // namespace caffe
