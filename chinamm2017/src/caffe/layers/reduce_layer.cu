#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/reduce_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReduceForward(const int count,
  const Dtype alpha, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = in[index] > alpha ? in[index] : 0;
  }
}

template <typename Dtype>
void ReduceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  ++iter_;
  alpha_ = this->blobs_[0]->cpu_data()[0];
  if(iter_ == 25)
  {
    LOG(ERROR)<<"reduce alpha : ******* "<<alpha_;
    iter_ = 0;
  }
  // LOG(INFO)<<alpha_;
  // int channels_ = bottom[0]->channels();
  // int batch_size_ = bottom[0]->num();
  const int count = bottom[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  // const int code_length_ = count / batch_size_;
  ReduceForward<Dtype><<<CAFFE_GET_BLOCKS(count),
    CAFFE_CUDA_NUM_THREADS>>>(count, alpha_,
    bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ReduceBackward(const int count, const Dtype alpha,
  const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, count) {
    out_diff[index] =
      in_data[index] > alpha ? in_diff[index] : 0;
  }
}

template <typename Dtype>
void ReduceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    ReduceBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, alpha_,
      top_diff, bottom_data, bottom_diff);
    Dtype sum(0);
    for (int i = 0; i < count; ++i)
    {
      if(top[0]->cpu_data()[i]>0)
      {
        ++sum;
      }
    }
    Dtype rho = 
      this->layer_param_.reduce_param().rho();
    sum /= count;
    // Dtype lr = 
    //   this->layer_param_.reduce_param().lr();
    alpha_ += sum * lr_ * ((1.-rho) / (1.-sum) - rho / sum);
    if (iter_ % 249 == 0)
    {
      LOG(ERROR)<<"this sparsity: "<<sum;
    }
    this->blobs_[0]->mutable_cpu_data()[0] = alpha_;
    this->blobs_[0]->mutable_cpu_diff()[0] = 0;
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReduceLayer);


}  // namespace caffe
