#include <algorithm>
#include <vector>
#include <map>

#include "caffe/layers/similarity3_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe
{
	template <typename Dtype>
	inline bool judge(int label_num, Dtype* label, int i, int j)
	{
		if (label_num > 1)
		{
			return caffe_cpu_dot(label_num,
				label + i * label_num,
				label + j * label_num) > 0;
		}
		else
		{
			return label[i] == label[j];
		}
	}

	template <typename Dtype>
	void Similarity3LossLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const int label_num = bottom[1]->count() / batch_size_;
		const int count = bottom[0]->count();
		const Dtype* label = bottom[1]->cpu_data();
		const Dtype loss_weight = top[0]->cpu_diff()[0];
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		Dtype loss(0.0);
		bool if_sim(0);

		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, batch_size_,
			batch_size_, code_length_, Dtype(1.0), bottom_data,
			bottom_data, Dtype(0.0),
			dot_product_mat_.mutable_gpu_data());

		// int temp(0);
		// caffe_gpu_max<Dtype>(count, dot_product_mat_.gpu_data(),
		// 	&temp);
		// if (dot_product_mat_.cpu_data()[temp] > 2.0)
		// {
		// 	LOG(FATAL) << "SimED2LossLayer need l2"
		// 		<<" normalization first!";
		// }

		// caffe_gpu_min<Dtype>(count, dot_product_mat_.gpu_data(),
		// 	&temp);
		// if (dot_product_mat_.cpu_data()[temp] < -2.0)
		// {
		// 	LOG(FATAL) << "SimED2LossLayer need l2"
		// 		<<" normalization first!";
		// }

		// bottom diff must be initialized !!!!!!!!!!!!!!!!!!!!
		hipMemset(bottom_diff, 0, sizeof(Dtype) * count);

		// srand((unsigned int)(time(NULL)));
		int num_par(0);
		for (int i = 0; i < batch_size_ - 1; ++i)
		{
			for (int j = i + 1; j < batch_size_; ++j)
			{
				if_sim = judge(label_num, label, i, j);
				if (!if_sim)
				{
					continue;
				}
				Dtype sim_sam =
					dot_product_mat_.cpu_data()[i * batch_size_ + j];
				for (int k = 0; k < batch_size_; ++k)
				{
					if_sim = judge(label_num, label, i, k);
					if (if_sim)
					{
						continue;
					}
					Dtype sim_dissam =
						dot_product_mat_.cpu_data()[i * batch_size_ + k];
					if (alpha_ + sim_dissam - sim_sam < 0)
					{
						continue; //this negative sample is too easy
					}
					if (beta_ + sim_dissam - sim_sam > 0)
					{
						continue; //this negative sample is too hard
					}
					loss += alpha_ + sim_dissam - sim_sam;
					++num_par;
					//gradient w.r.t i
					caffe_sub<Dtype>(code_length_,
						bottom[0]->cpu_data() + k * code_length_,
						bottom[0]->cpu_data() + j * code_length_,
						temp_diff_vec_.mutable_cpu_data());
					caffe_cpu_axpby<Dtype>(code_length_,
						loss_weight, temp_diff_vec_.cpu_data(), 1,
						bottom[0]->mutable_cpu_diff() + i * code_length_);
					//gradient w.r.t j
					caffe_cpu_axpby<Dtype>(code_length_,
						-loss_weight,
						bottom[0]->cpu_data() + i * code_length_, 1,
						bottom[0]->mutable_cpu_diff() + j * code_length_);
					//gradient w.r.t k
					caffe_cpu_axpby<Dtype>(code_length_,
						loss_weight,
						bottom[0]->cpu_data() + i * code_length_, 1,
						bottom[0]->mutable_cpu_diff() + k * code_length_);
				}
			}
		}
		caffe_gpu_scal<Dtype>(count, 1.0 / (num_par + 1e-5),
			bottom_diff);
		top[0]->mutable_cpu_data()[0] = loss / (num_par + 1e-5);
		// LOG(INFO)<<"number of hard pairs: "<<num_par;
	}

	template <typename Dtype>
	void Similarity3LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
	{
		if (propagate_down[1])
		{
			LOG(FATAL) << this->type() << " Layer cannot backpropagate to label inputs. ";
		}
	}
	INSTANTIATE_LAYER_GPU_FUNCS(Similarity3LossLayer);
}  // namespace caffe



// int sim_j(-1);
// int dis_j(-1);
// Dtype similarity_sim(1.1);
// // Dtype similarity_dis(0);
// Dtype sub_loss(-1);
// for (int j = 0; j < batch_size_; ++j)
// {
// 	if_sim = judge(label_num, label, i, j);
// 	if (!if_sim)
// 	{
// 		continue;
// 	}
// 	int locat_temp = i * batch_size_ + j;
// 	Dtype similarity_temp = 
// 		dot_product_mat_.cpu_data()[locat_temp];
//   if (similarity_temp == 0)
//   {
//   	continue;
//   }
//   if (similarity_temp >= similarity_sim)
//   {
//   	continue;
//   }
//   similarity_sim = similarity_temp;
//   sim_j = j;
// }
// for (int j = 0; j < batch_size_; ++j)
// {
// 	if_sim = judge(label_num, label, i, j);
// 	if (if_sim)
// 	{
// 		continue;
// 	}
// 	int locat_temp = i * batch_size_ + j;
// 	Dtype similarity_temp =
// 		dot_product_mat_.cpu_data()[locat_temp];
// 	// if (similarity_temp == 0)
// 	// {
// 	// 	continue;
// 	// }
// 	Dtype loss_temp(0);
// 	if(imagenetsim_ > 0)
// 	{
// 		loss_temp = similarity_temp - similarity_sim
// 			+ alpha_/(1+sim_mat_[locat_temp])/(1+sim_mat_[locat_temp]);
// 	}
// 	else
// 	{
// 		loss_temp = similarity_temp + alpha_ - similarity_sim;
// 	}
// 	if (loss_temp <= sub_loss)
// 	{
// 		continue;
// 	}
// 	// similarity_dis = similarity_temp;
// 	sub_loss = loss_temp;
// 	// LOG(INFO)<<similarity_dis<<"aaa"<<label[i]<<"ad"<<label[j];

// 	dis_j = j;
// }
// ++num_par;
// if ( sub_loss > 0)
// {
// 	if (dis_j == -1)
// 	{
// 		continue;
// 	}
// 	if (sim_j == -1)
// 	{
// 		continue;
// 	}
// 	loss += sub_loss;
// 	// gradient w.r.t i
// 	caffe_gpu_axpby<Dtype>(code_length_, loss_weight,
// 		bottom_data + dis_j * code_length_, 1,
// 		bottom_diff + i * code_length_);

// 	caffe_gpu_axpby<Dtype>(code_length_, -loss_weight,
// 		bottom_data + sim_j * code_length_, 1,
// 		bottom_diff + i * code_length_);

// 	// gradient w.r.t dis_j
// 	caffe_gpu_axpby<Dtype>(code_length_, loss_weight,
// 		bottom_data + i * code_length_, 1,
// 		bottom_diff + dis_j * code_length_);

// 	// gradient w.r.t sim_j
// 	caffe_gpu_axpby<Dtype>(code_length_, -loss_weight,
// 		bottom_data + i * code_length_, 1,
// 		bottom_diff + sim_j * code_length_);