#include <algorithm>
#include <vector>
#include <map>

#include "caffe/layers/similarity2_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe
{
	template <typename Dtype>
	inline bool judge(int label_num, Dtype* label, int i, int j)
	{
		if (label_num > 1)
		{
			return caffe_cpu_dot(label_num,
				label + i * label_num,
				label + j * label_num) > 0;
		}
		else
		{
			return label[i] == label[j];
		}
	}

	template <typename Dtype>
	void Similarity2LossLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const int label_num = bottom[1]->count() / batch_size_;
		const int count = bottom[0]->count();
		const Dtype* label = bottom[1]->cpu_data();
		const Dtype loss_weight = top[0]->cpu_diff()[0] * 2.0 /
			(Dtype)batch_size_ / (Dtype)(batch_size_ - 1);
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		Dtype loss(0.0);
		bool if_sim(0);

		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, batch_size_,
			batch_size_, code_length_, Dtype(1.0), bottom_data,
			bottom_data, Dtype(0.0),
			dot_product_mat_.mutable_gpu_data());

		// int temp(0);
		// caffe_gpu_max<Dtype>(count, dot_product_mat_.gpu_data(),
		// 	&temp);
		// if (dot_product_mat_.cpu_data()[temp] > 2.0)
		// {
		// 	LOG(FATAL) << "SimED2LossLayer need l2"
		// 		<<" normalization first!";
		// }

		// caffe_gpu_min<Dtype>(count, dot_product_mat_.gpu_data(),
		// 	&temp);
		// if (dot_product_mat_.cpu_data()[temp] < -2.0)
		// {
		// 	LOG(FATAL) << "SimED2LossLayer need l2"
		// 		<<" normalization first!";
		// }

		// bottom diff must be initialized !!!!!!!!!!!!!!!!!!!!
		hipMemset(bottom_diff, 0, sizeof(Dtype) * count);

		// srand((unsigned int)(time(NULL)));
		for (int i = 0; i < batch_size_ - 1; ++i)
		{
			for (int j = i + 1; j < batch_size_; ++j)
			{
				if_sim = judge(label_num, label, i, j);
				Dtype sub_loss(0.0);
				if (if_sim)
				{
					sub_loss = alpha_ - 
						dot_product_mat_.cpu_data()[i * batch_size_ + j];
				}
				else
				{
					sub_loss =
						dot_product_mat_.cpu_data()[i * batch_size_ + j];
				}
				if (sub_loss > 0)
				{
					loss += sub_loss;
					int factor = 1 - (int)if_sim * 2;
					//gradient w.r.t i
					caffe_cpu_axpby<Dtype>(code_length_,
						loss_weight * factor,
						bottom[0]->cpu_data() + j * code_length_,
						Dtype(1.0),
						bottom[0]->mutable_cpu_diff() + i * code_length_);
					//gradient w.r.t j
					caffe_cpu_axpby<Dtype>(code_length_,
						loss_weight * factor,
						bottom[0]->cpu_data() + i * code_length_,
						Dtype(1.0),
						bottom[0]->mutable_cpu_diff() + j * code_length_);
				}
			}
		}
		top[0]->mutable_cpu_data()[0] = loss  * 2.0 /
			(Dtype)batch_size_ / (Dtype)(batch_size_ - 1);
	}

	template <typename Dtype>
	void Similarity2LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom)
	{
		if (propagate_down[1])
		{
			LOG(FATAL) << this->type() << " Layer cannot backpropagate to label inputs. ";
		}
	}
	INSTANTIATE_LAYER_GPU_FUNCS(Similarity2LossLayer);
}  // namespace caffe