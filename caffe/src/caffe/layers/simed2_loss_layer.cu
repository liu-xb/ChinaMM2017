#include <algorithm>
#include <vector>
#include <map>

#include "caffe/layers/simed2_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe
{
	template <typename Dtype>
	inline bool judge(int label_num, Dtype* label, int i, int j)
	{
		if (label_num > 1)
		{
			return caffe_cpu_dot(label_num,
				label + i * label_num,
				label + j * label_num) > 0;
		}
		else
		{
			return label[i] == label[j];
		}
	}

	template <typename Dtype>
	void SimED2LossLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const int label_num = bottom[1]->count() / batch_size_;
		const int count = bottom[0]->count();
		const Dtype* label = bottom[1]->cpu_data();
		const Dtype loss_weight =	1.0 * top[0]->cpu_diff()[0] /
			batch_size_ / (batch_size_ - 1) * 2;
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		Dtype loss(0.0);
		bool if_sim(0);

		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, batch_size_,
			batch_size_, code_length_, Dtype(1.0), bottom_data,
			bottom_data, Dtype(0.0),
			dot_product_mat_.mutable_gpu_data());

		// int temp(0);
		// caffe_gpu_max<Dtype>(count, dot_product_mat_.gpu_data(),
		// 	&temp);
		// if (dot_product_mat_.cpu_data()[temp] > 2.0)
		// {
		// 	LOG(FATAL) << "SimED2LossLayer need l2"
		// 		<<" normalization first!";
		// }

		// caffe_gpu_min<Dtype>(count, dot_product_mat_.gpu_data(),
		// 	&temp);
		// if (dot_product_mat_.cpu_data()[temp] < -2.0)
		// {
		// 	LOG(FATAL) << "SimED2LossLayer need l2"
		// 		<<" normalization first!";
		// }

		//!!!!bottom diff must be initialized!!!!!!!!!!
		hipMemset(bottom_diff, 0, sizeof(Dtype) * count);
		// srand((unsigned int)(time(NULL)));
		// int num_par(0);

		for (int i = 0; i < batch_size_ - 1; ++i)
		{
			for (int j = i + 1; j < batch_size_; ++j)
			{
				Dtype sub_loss(0.0);
				if_sim = judge(label_num, label, i, j);
				if (if_sim)
				{
					sub_loss = 2 - 
						2 * dot_product_mat_.cpu_data()[i * batch_size_ + j];
				}
				else
				{
					sub_loss = alpha_ - 2 + 
						2 * dot_product_mat_.cpu_data()[i * batch_size_ + j];
				}
				if (sub_loss > 0)
				{// whether we need to compute the gradient
					loss += sub_loss;
					int factor = (int)if_sim * 2 - 1;
					caffe_sub<Dtype>(code_length_,
						bottom[0]->cpu_data() + i * code_length_,
						bottom[0]->cpu_data() + j * code_length_,
						temp_diff_vec_.mutable_cpu_data());
					// gradient with respect to i
					caffe_cpu_axpby<Dtype>(code_length_,
						loss_weight * factor, temp_diff_vec_.cpu_data(),
						Dtype(1.0),
						bottom[0]->mutable_cpu_diff() + i * code_length_);
					// gradient with respect to j
					caffe_cpu_axpby<Dtype>(code_length_,
						-loss_weight * factor, temp_diff_vec_.cpu_data(),
						Dtype(1),
						bottom[0]->mutable_cpu_diff() + j * code_length_);
				}
			}
		}
		top[0]->mutable_cpu_data()[0] =
			loss / batch_size_ / (batch_size_ - 1) * 2;
	}

	template <typename Dtype>
	void SimED2LossLayer<Dtype>::Backward_gpu(
		const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom)
	{
		if (propagate_down[1])
		{
			LOG(FATAL) << this->type() <<
				" Layer cannot backpropagate to label inputs. ";
		}
	}
	INSTANTIATE_LAYER_GPU_FUNCS(SimED2LossLayer);
}  // namespace caffe


// if (label_num > 1)
				// {
				// 	if_sim = caffe_cpu_dot(label_num, 
				// 		label + i * label_num,
				// 		label + j * label_num) > 0;
				// }
				// else
				// {
				// 	if_sim = ((static_cast<int>(label[i])) == 
				// 		(static_cast<int>(label[j])));
				// }