#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/binarize1_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Binarize1Forward(const int count,
  const Dtype alpha, const Dtype* in, Dtype* out,
  Dtype maxvalue, Dtype minvalue) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = in[index] > alpha ? maxvalue : minvalue;
  }
}

template <typename Dtype>
void Binarize1Layer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  ++iter_;
  alpha_ = this->blobs_[0]->cpu_data()[0];
  if(iter_ == 250)
  {
    LOG(INFO)<<"reduce alpha : ---"<<alpha_;
    iter_ = 0;
  }
  const int count = bottom[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Binarize1Forward<Dtype><<<CAFFE_GET_BLOCKS(count),
    CAFFE_CUDA_NUM_THREADS>>>(count, alpha_,
      bottom_data, top_data, maxvalue_, minvalue_);
  CUDA_POST_KERNEL_CHECK;
}

// template <typename Dtype>
// __global__ void ReduceBackward(const int count, const Dtype alpha,
//   const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff) {
//   CUDA_KERNEL_LOOP(index, count) {
//     out_diff[index] =
//       in_data[index] > alpha ? in_diff[index] : 0;
//   }
// }

template <typename Dtype>
void Binarize1Layer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* top_data = top[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    vector<int> temp_diff_shape(1, count);
    temp_diff_.Reshape(temp_diff_shape);
    // ReduceBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
    //   CAFFE_CUDA_NUM_THREADS>>>(count, alpha_,
    //   top_diff, bottom_data, bottom_diff);
    caffe_gpu_memcpy(count * sizeof(Dtype),
      top_diff, bottom_diff);
    // Dtype sum(0);
    // for (int i = 0; i < count; ++i)
    // {
    //   if(top[0]->cpu_data()[i]>0)
    //   {
    //     ++sum;
    //   }
    // }
    // Dtype rho = 
    //   this->layer_param_.reduce_param().rho();
    // sum /= count;
    // Dtype lr = 
    //   this->layer_param_.reduce_param().lr();
    // alpha_ += sum * lr * ((1.-rho) / (1.-sum) - rho / sum);
    // this->blobs_[0]->mutable_cpu_data()[0] = alpha_;
    Dtype temp(0);
    caffe_gpu_asum<Dtype>(count, bottom_diff, &temp);
    caffe_gpu_sub(count, bottom_data, top_data, temp_diff_.mutable_gpu_data());
    caffe_gpu_axpy(count, 1 / count * tradeoff,
      temp_diff_.gpu_data(), bottom_diff);
    this->blobs_[0]->mutable_cpu_diff()[0] = - temp / (Dtype)count;
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(Binarize1Layer);


}  // namespace caffe
