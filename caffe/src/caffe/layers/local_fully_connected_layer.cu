#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/local_fully_connected_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void LocalFullyConnectedLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();

  const Dtype** bottom_i = NULL;
  Dtype** top_i = NULL;
  const Dtype** weight_i = NULL;
  size_t size = C_ * sizeof(Dtype*);
  hipMalloc(&bottom_i, size);
  hipMalloc(&top_i, size);
  hipMalloc(&weight_i, size);

  Dtype* alpha = new Dtype [C_];
  Dtype* beta = new Dtype [C_];
  const Dtype** bottom_host = new const Dtype* [C_];
  Dtype** top_host = new Dtype* [C_];
  const Dtype ** weight_host = new const Dtype* [C_];

  for ( int i = 0; i < C_; ++i) {
	bottom_host [i] = bottom_data + K_ * i;
	top_host [i] = top_data + N_ * i;
	weight_host [i] = weight + N_ * K_ * i;
	alpha [i] = (Dtype)1.0;
	beta [i] = (Dtype)0.0;
  }

  CUDA_CHECK(hipMemcpy(bottom_i, bottom_host, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(weight_i, weight_host, size, hipMemcpyHostToDevice));
  hipMemcpy(top_i, top_host, size, hipMemcpyHostToDevice);
  lfcl_gpu_gemmBatched_forward(M_, N_, K_, C_, alpha, bottom_i, weight_i, beta, top_i);

  hipFree(bottom_i);
  hipFree(top_i);
  hipFree(weight_i);
  delete alpha;
  delete beta;
  delete bottom_host;
  delete top_host;
  delete weight_host;
  //for ( int i = 0; i < C_; ++i){
  //        const Dtype* bottom_data_i = bottom_data + K_ * i;
  //        const Dtype* weight_i  = weight + N_ * K_ * i;
  //        Dtype* top_data_i = top_data + N_ * i;
  //        lfcl_gpu_gemm<Dtype>(M_, N_, K_, C_, (Dtype)1.0,
  //               bottom_data_i, weight_i, (Dtype)0.0, top_data_i);
  //}

  if (bias_term_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, C_ * N_, 1,
              (Dtype)1.0, bias_multiplier_.gpu_data(),
              this->blobs_[1]->gpu_data(), (Dtype)1.0, top_data);
  }
/*const Dtype* bottom_data = bottom[0]->cpu_data();
  Dtype* top_data = top[0]->mutable_cpu_data();
  const Dtype* weight = this->blobs_[0]->cpu_data();
  for ( int i = 0; i < C_; ++i){
          const Dtype* bottom_data_ij = bottom_data + K_ * i;
          const Dtype* weight_j  = weight + N_ * K_ * i;
          Dtype* top_data_ij = top_data + N_ * i;
          lfcl_cpu_gemm<Dtype>(M_, N_, K_, C_, (Dtype)1.0,
                  bottom_data_ij, weight_j, (Dtype)0.0, top_data_ij);
  }*/
}

template <typename Dtype>
void LocalFullyConnectedLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {

    if (this->param_propagate_down_[0]) {
        Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
        const Dtype* top_diff = top[0]->gpu_diff();
        const Dtype* bottom_data = bottom[0]->gpu_data();
	
        // Gradient with respect to weight
	const Dtype** bottom_i = NULL;
 	const Dtype** top_i = NULL;
  	Dtype** weight_i = NULL;
  	size_t size = C_ * sizeof(Dtype*);
  	hipMalloc(&bottom_i, size);
  	hipMalloc(&top_i, size);
  	hipMalloc(&weight_i, size);

  	Dtype* alpha = new Dtype [C_];
  	//Dtype* beta = new Dtype [C_];
  	const Dtype** bottom_host = new const Dtype* [C_];
  	const Dtype** top_host = new const Dtype* [C_];
  	Dtype ** weight_host = new Dtype* [C_];

  	for ( int i = 0; i < C_; ++i) {
	    bottom_host [i] = bottom_data + K_ * i;
	    top_host [i] = top_diff + N_ * i;
	    weight_host [i] = weight_diff + N_ * K_ * i;
	    alpha [i] = (Dtype)1.0;
	    //beta [i] = (Dtype)0.0;
  	}

  	hipMemcpy(bottom_i, bottom_host, size, hipMemcpyHostToDevice);
  	hipMemcpy(weight_i, weight_host, size, hipMemcpyHostToDevice);
  	hipMemcpy(top_i, top_host, size, hipMemcpyHostToDevice);
  	lfcl_gpu_gemmBatched_backward_weight(M_, N_, K_, C_, alpha, top_i, bottom_i, alpha, weight_i);

  	hipFree(bottom_i);
  	hipFree(top_i);
  	hipFree(weight_i);
  	delete alpha;
//  	delete beta;
  	delete bottom_host;
  	delete top_host;
  	delete weight_host;
       /* for (int i = 0; i < M_;  ++i) {
            for (int j = 0; j < C_; ++j) {
                const Dtype* top_diff_ij = top_diff + N_ * C_ * i + N_ * j;
                const Dtype* bottom_data_ij = 
                        bottom_data + C_ * K_ * i + K_ * j;
                Dtype* weight_diff_j = weight_diff + N_ * K_ * j;
                caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, N_, K_, 1, 
                        (Dtype)1.0, top_diff_ij, bottom_data_ij, (Dtype)1.0,
                        weight_diff_j);
            }
        }*/
    }
    if (bias_term_ && this->param_propagate_down_[1]) {
   	const Dtype* top_diff = top[0]->gpu_diff();
   	// Gradient with respect to bias
    	caffe_gpu_gemv<Dtype>(CblasTrans, M_, C_ * N_, (Dtype)1., top_diff,
        bias_multiplier_.gpu_data(), (Dtype)1.,
        this->blobs_[1]->mutable_gpu_diff());
    }
    if (propagate_down[0]) {
        // Gradient with respect to bottom data
        const Dtype* top_diff = top[0]->gpu_diff();
        const Dtype* weight = this->blobs_[0]->gpu_data();
        Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
//////////
	Dtype** bottom_i = NULL;
 	const Dtype** top_i = NULL;
  	const Dtype** weight_i = NULL;
  	size_t size = C_ * sizeof(Dtype*);
  	hipMalloc(&bottom_i, size);
  	hipMalloc(&top_i, size);
  	hipMalloc(&weight_i, size);

  	Dtype* alpha = new Dtype [C_];
  	Dtype* beta = new Dtype [C_];
  	Dtype** bottom_host = new Dtype* [C_];
  	const Dtype** top_host = new const Dtype* [C_];
  	const Dtype ** weight_host = new const Dtype* [C_];

  	for ( int i = 0; i < C_; ++i) {
	    bottom_host [i] = bottom_diff + K_ * i;
	    top_host [i] = top_diff + N_ * i;
	    weight_host [i] = weight + N_ * K_ * i;
	    alpha [i] = (Dtype)1.0;
	    beta [i] = (Dtype)0.0;
  	}

  	hipMemcpy(bottom_i, bottom_host, size, hipMemcpyHostToDevice);
  	hipMemcpy(weight_i, weight_host, size, hipMemcpyHostToDevice);
  	hipMemcpy(top_i, top_host, size, hipMemcpyHostToDevice);
  	lfcl_gpu_gemmBatched_backward_bottom(M_, N_, K_, C_, alpha, weight_i, top_i, beta, bottom_i);

  	hipFree(bottom_i);
  	hipFree(top_i);
  	hipFree(weight_i);
  	delete alpha;
  	delete beta;
  	delete bottom_host;
  	delete top_host;
  	delete weight_host;
        /*for(int i = 0; i < M_; ++i) {
            for(int j = 0; j < C_; ++j) {
                const Dtype* top_diff_ij = top_diff + N_ * C_ * i + N_ * j;
                const Dtype* weight_j = weight + N_ * K_ * j;
                Dtype* bottom_diff_ij = bottom_diff + C_ * K_ * i + K_ * j;
                caffe_gpu_gemm<Dtype>(
                        CblasNoTrans, CblasTrans, 1, K_, N_, (Dtype)1.0, 
                        top_diff_ij, weight_j, (Dtype)0.0, bottom_diff_ij);
            }
        }*/
    }
}

INSTANTIATE_LAYER_GPU_FUNCS(LocalFullyConnectedLayer);

}  // namespace caffe
