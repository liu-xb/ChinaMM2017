#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/uniform_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe
{
  template <typename Dtype>
  __global__  void UniformLossForward(const int n, const Dtype* in,
    Dtype* out)
  {
    CUDA_KERNEL_LOOP(index, n)
    {
      out[index] = in[index] > (Dtype)0.0;
    }
  }

  template <typename Dtype>
  __global__  void UniformLossBackward(const int n, const Dtype* in,
    Dtype* out)
  {
    CUDA_KERNEL_LOOP(index, n)
    {
      out[index] = in[index];
    }
  }

  template <typename Dtype>
  void UniformLossLayer<Dtype>::Forward_gpu(
  const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top)
  {
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    const int count = bottom[0]->count();
    if (loss_weight == 0)
    {
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      hipMemset(bottom_diff, 0, sizeof(Dtype) * count);
      top[0]->mutable_cpu_data()[0] = 0;
    }
    else
    {
      ++iter_;
      const Dtype* bottom_data = bottom[0]->gpu_data();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      UniformLossForward<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, bottom_data, bottom_diff);
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1,
        code_length_, batch_size_, (Dtype)1.0, vec_sum_.gpu_data(),
        bottom_diff, (Dtype)0.0, temp_code_.mutable_gpu_data());

      caffe_gpu_axpy<Dtype>(code_length_,
        (Dtype)(1.0 / (Dtype)batch_size_ / (Dtype)STEP_SIZE_),
        temp_code_.gpu_data(), current_code_.mutable_gpu_data());
      if (iter_ < STEP_SIZE_)
      {
        top[0]->mutable_cpu_data()[0] = loss_;
        UniformLossBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
          CAFFE_CUDA_NUM_THREADS>>>(count, current_diff_.gpu_data(),
            bottom_diff);
      }
      else
      {
        iter_ = 0;
        Dtype loss(0);
        Dtype sum_grad = 0;
        for(int i = 0; i < code_length_; ++i)
        {
          Dtype rho = current_code_.cpu_data()[i];
          rho = rho >= 1 ? 0.999 : rho;
          rho = rho <= 0 ? 1e-20 : rho;
          loss += alpha_ * log(alpha_ / rho) +
            (1 - alpha_) * log((1 - alpha_) / (1 - rho));
          Dtype grad = (1 - alpha_) / (1 - rho) -
            alpha_ / rho;
          sum_grad += grad * grad;
          for (int j = 0; j < batch_size_; ++j)
          {
            bottom[0]->mutable_cpu_diff()[i + j * code_length_] = grad;
          }
        }
        sum_grad = pow(sum_grad, 0.5);
        caffe_gpu_scale<Dtype>(count, loss_weight / sum_grad, bottom_diff,
          current_diff_.mutable_gpu_data());
        UniformLossBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
          CAFFE_CUDA_NUM_THREADS>>>(count, current_diff_.gpu_data(),
          bottom_diff);

        hipMemset(current_code_.mutable_gpu_data(), 0,
          sizeof(Dtype) * code_length_);
        loss_ = loss / code_length_;
        top[0]->mutable_cpu_data()[0] = loss_;
      }
    }
  }

  template <typename Dtype>
  void UniformLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
  const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) 
  {
    if (propagate_down[1])
      {
        LOG(FATAL) << this->type()
             << " Layer cannot backpropagate to label inputs.";
      }
  }

  INSTANTIATE_LAYER_GPU_FUNCS(UniformLossLayer);

}  // namespace caffe