#include <algorithm>
#include <vector>
#include <map>

#include "caffe/layers/simed3_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe
{
	template <typename Dtype>
	inline bool judge(int label_num, Dtype* label, int i, int j)
	{
		if (label_num > 1)
		{
			return caffe_cpu_dot(label_num,
				label + i * label_num,
				label + j * label_num) > 0;
		}
		else
		{
			return label[i] == label[j];
		}
	}

	template <typename Dtype>
	void SimED3LossLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top)
	{
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const int label_num = bottom[1]->count() / batch_size_;
		const int count = bottom[0]->count();
		const Dtype* label = bottom[1]->cpu_data();
		const Dtype loss_weight = top[0]->cpu_diff()[0];
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		Dtype loss(0.0);
		bool if_sim(0);

		caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, batch_size_,
			batch_size_, code_length_, Dtype(1.0), bottom_data,
			bottom_data, Dtype(0.0),
			dot_product_mat_.mutable_gpu_data());

		// int temp(0);
		// caffe_gpu_max<Dtype>(count, dot_product_mat_.gpu_data(),
		// 	&temp);
		// if (dot_product_mat_.cpu_data()[temp] > 2.0)
		// {
		// 	LOG(FATAL) << "SimED2LossLayer need l2"
		// 		<<" normalization first!";
		// }

		// caffe_gpu_min<Dtype>(count, dot_product_mat_.gpu_data(),
		// 	&temp);
		// if (dot_product_mat_.cpu_data()[temp] < -2.0)
		// {
		// 	LOG(FATAL) << "SimED2LossLayer need l2"
		// 		<<" normalization first!";
		// }

		//!!!!bottom diff has to be initialized!!!!!!!!!!!!!!!!!!!!
		hipMemset(bottom_diff, 0, sizeof(Dtype) * count);

		// srand((unsigned int)(time(NULL)));
		int num_par(0);

		for (int i = 0; i < batch_size_ - 1; ++i)
		{
			for (int j = i + 1; j < batch_size_; ++j)
			{
				// Dtype sub_loss(0.0);
				if_sim = judge(label_num, label, i, j);
				if (!if_sim)
				{
					continue;
				}
				Dtype sim_sam = 
					dot_product_mat_.cpu_data()[i * batch_size_ + j];
				for (int k = 0; k < batch_size_; ++k)
				{
					if_sim = judge(label_num, label, i, k);
					if (if_sim)
					{
						continue;
					}
					Dtype sim_dissam =
						dot_product_mat_.cpu_data()[i * batch_size_ + k];
					if (alpha_ * 0.5 + sim_dissam - sim_sam < 0)
					{
						continue; //this negative sample is too easy
					}
					if (beta_ * 0.5 + sim_dissam - sim_sam > 0)
					{
						continue; //this negative sample is too hard
					}
					loss += alpha_ + 2 * (sim_dissam - sim_sam);
					++num_par;
					//gradient w.r.t. i
					caffe_sub<Dtype>(code_length_,
						bottom[0]->cpu_data() + k * code_length_,
						bottom[0]->cpu_data() + j * code_length_,
						temp_diff_vec_.mutable_cpu_data());
					caffe_cpu_axpby<Dtype>(code_length_,
						loss_weight, temp_diff_vec_.cpu_data(), 1,
						bottom[0]->mutable_cpu_diff() + i * code_length_);
					//gradient w.r.t. j
					caffe_sub<Dtype>(code_length_,
						bottom[0]->cpu_data() + j * code_length_,
						bottom[0]->cpu_data() + i * code_length_,
						temp_diff_vec_.mutable_cpu_data());
					caffe_cpu_axpby<Dtype>(code_length_,
						loss_weight, temp_diff_vec_.cpu_data(), 1,
						bottom[0]->mutable_cpu_diff() + j * code_length_);
					//gradient w.r.t k
					caffe_sub<Dtype>(code_length_,
						bottom[0]->cpu_data() + i * code_length_,
						bottom[0]->cpu_data() + k * code_length_,
						temp_diff_vec_.mutable_cpu_data());
					caffe_cpu_axpby<Dtype>(code_length_,
						loss_weight, temp_diff_vec_.cpu_data(), 1,
						bottom[0]->mutable_cpu_diff() + k * code_length_);
				}
			}
		}
		// LOG(INFO)<<"dadf"<<num_par;
		caffe_gpu_scal<Dtype>(count, 1.0 / (num_par + 1e-5), bottom_diff);
		top[0]->mutable_cpu_data()[0] = loss / (num_par + 1e-5);
	}

	template <typename Dtype>
	void SimED3LossLayer<Dtype>::Backward_gpu(
		const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down,
		const vector<Blob<Dtype>*>& bottom)
	{
		if (propagate_down[1])
		{
			LOG(FATAL) << this->type() <<
				" Layer cannot backpropagate to label inputs. ";
		}
	}
	INSTANTIATE_LAYER_GPU_FUNCS(SimED3LossLayer);
}  // namespace caffe