#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/sparsity_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SparsityLossForward(const int n, const Dtype* in, Dtype* out, const Dtype value)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    out[index] = in[index] > 0 ? value : 0.0;
  }
}

template <typename Dtype>
__global__ void SparsityLossBackward(const int n, Dtype* out, const Dtype alpha)
{
  CUDA_KERNEL_LOOP(index, n)
  {
    out[index] = out[index] * alpha;
  }
}

template <typename Dtype>
void SparsityLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top)
{
  const Dtype loss_weight = top[0]->cpu_diff()[0];
  if (loss_weight == 0)
  {
    top[0]->mutable_cpu_data()[0] = 0;
  }
  else
  {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const int count = bottom[0]->count();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype loss(0.0);
    SparsityLossForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom_diff, loss_weight);
    caffe_gpu_asum(count, bottom_diff, &loss);
    loss /= (Dtype)count * loss_weight;
    if (loss < alpha_) 
    {
      loss = 0;
      hipMemset(bottom_diff, 0., count);
    }
    else
    {
      loss -= alpha_;
      SparsityLossBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_diff, loss);
    }
    top[0]->mutable_cpu_data()[0] = loss;
  }
}

template <typename Dtype>
void SparsityLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) 
  {
    if (propagate_down[1])
      {
        LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
      }
  }

INSTANTIATE_LAYER_GPU_FUNCS(SparsityLossLayer);

}  // namespace caffe
