#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/convrelu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ConvReLUForward(const int channels, const int len, 
  const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, channels) {
    Dtype sum(0);
    for (int i = 0; i < len; ++i)
    {
      sum += in[index * len + i];
    }
    for (int i = 0; i < len; ++i)
    {
      out[i + index * len] = sum > 0 ? in[i + index * len] : 0;
    }
  }
}

template <typename Dtype>
void ConvReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int channels_ = bottom[0]->channels();
  int batch_size_ = bottom[0]->num();
  const int count = bottom[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int code_length_ = count / batch_size_;
  for (int i = 0; i < batch_size_; ++i)
  {
    ConvReLUForward<Dtype><<<CAFFE_GET_BLOCKS(channels_),
      CAFFE_CUDA_NUM_THREADS>>>(channels_, count / batch_size_ / channels_,
      bottom_data + i * code_length_, top_data + i * code_length_);
  }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ConvReLUBackward(const int channels, const int len,
  const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, channels) {
    Dtype sum(0);
    for (int i = 0; i < len; ++i)
    {
      sum += in_data[i + index * len];
    }
    for (int i = 0; i < len; ++i)
    {
      out_diff[i + index * len] = sum > 0 ? in_diff[i + index * len] : 0;
    }
  }
}

template <typename Dtype>
void ConvReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    int channels_ = bottom[0]->channels();
    int batch_size_ = bottom[0]->num();
    const int code_length_ = count / batch_size_;
    for (int i = 0; i < batch_size_; ++i)
    {
      ConvReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(channels_),
        CAFFE_CUDA_NUM_THREADS>>>(channels_, count / batch_size_ / channels_,
        top_diff + i * code_length_, bottom_data + i * code_length_,
        bottom_diff + i * code_length_);
    }
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ConvReLULayer);


}  // namespace caffe
